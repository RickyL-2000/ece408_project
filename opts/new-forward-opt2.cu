#include "hip/hip_runtime.h"
// Using Streams to overlap computation with data transfer

#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

#define BLOCK_WIDTH 16
#define TILE_WIDTH 16
#define CONV_DEBUG

#define TEST_NAME "opt2 10000"

__global__ void conv_forward_kernel(
    float *y, const float *x, const float *k, 
    const int B, const int M, const int C, const int H, const int W, const int K)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.

    Function paramter definitions:
    y - output
    x - input
    k - kernel
    B - batch_size (number of images in x)
    M - number of output feature maps
    C - number of input feature maps
    H - input height dimension
    W - input width dimension
    K - kernel height and width (K x K)
    */

    const int H_out = H - K + 1;
    const int W_out = W - K + 1;
    // (void)H_out; // silence declared but never referenced warning. remove this line when you start working
    // (void)W_out; // silence declared but never referenced warning. remove this line when you start working

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = y4d(0,0,0,0)
    // y4d(0,0,0,0) = a

#define y4d(i3, i2, i1, i0) y[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
#define x4d(i3, i2, i1, i0) x[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
#define k4d(i3, i2, i1, i0) k[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

    // Insert your GPU convolution kernel code here

    int W_num = ceil(W_out / (BLOCK_WIDTH * 1.0));
        // H_num = ceil(H_out / (BLOCK_WIDTH * 1.0));
    int b = blockIdx.x, m = blockIdx.y;
    int h = (blockIdx.z / W_num) * BLOCK_WIDTH + threadIdx.x,
        w = (blockIdx.z % W_num) * BLOCK_WIDTH + threadIdx.y;
    // int h = threadIdx.y, w = threadIdx.x;

    int c, p, q;
    float res = 0.0f;
    if (w >= W_out || h >= H_out) return;
    for (c = 0; c < C; ++c) {
        for (p = 0; p < K; ++p) {
            for (q = 0; q < K; ++q) {
                res += x4d(b, c, h+p, w+q) * k4d(m, c, p, q);
            }
        }
    }
    y4d(b, m, h, w) = res;

#undef y4d
#undef x4d
#undef k4d
}

	
__host__ void GPUInterface::conv_forward_gpu_prolog(
    float *host_y, const float *host_x, const float *host_k, 
    float **device_y_ptr, float **device_x_ptr, float **device_k_ptr, 
    const int B, const int M, const int C, const int H, const int W, const int K)
{
    // Allocate memory and copy over the relevant data structures to the GPU

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }

    const int nStreams = 16;

    const int H_out = H - K + 1;
    const int W_out = W - K + 1;

    const int y_bytes_s = B*M*H_out*W_out/nStreams, 
              x_bytes_s = B*C*H*W/nStreams, 
              y_bytes = B*M*H_out*W_out, 
              x_bytes = B*C*H*W, 
              k_bytes = M*C*K*K;

    const int y_size_s = y_bytes_s*sizeof(float), 
              x_size_s = x_bytes_s*sizeof(float), 
              y_size = y_bytes*sizeof(float), 
              x_size = x_bytes*sizeof(float), 
              k_size = k_bytes*sizeof(float);

    hipMalloc((void**) device_y_ptr, y_size);
    hipMalloc((void**) device_x_ptr, x_size);
    hipMalloc((void**) device_k_ptr, k_size);

    // configure stream
    int i;

    hipStream_t streams[nStreams];
    for (i = 0; i< nStreams; ++i) hipStreamCreate(&streams[i]);
    
    for (i = 0; i < nStreams; ++i) {
        hipMemcpyAsync((*device_x_ptr)+i*x_bytes_s, host_x+i*x_bytes_s, x_size_s, hipMemcpyHostToDevice, streams[i]);
    }
    // kernel should be used entirely, so no split
    hipMemcpyAsync(*device_k_ptr, host_k, k_size, hipMemcpyHostToDevice, streams[0]);
    
    // Now move the whole [conv_forward_gpu] here, merge them into a stream version
    const int W_num = ceil(W_out / (BLOCK_WIDTH * 1.0)),
              H_num = ceil(H_out / (BLOCK_WIDTH * 1.0));

    std::cout << "TEST NAME: " << TEST_NAME << std::endl;

#ifdef CONV_DEBUG
    // print dimension information
    std::cout << "Grid Dimension: " << B << " x " << M << " x " << W_num * H_num << std::endl;
    std::cout << "Block Dimension: " << BLOCK_WIDTH << " x " << BLOCK_WIDTH << " x " << 1 << std::endl;
    std::cout << "Kernel Dimension: " << M << " x " << C << " x " << K << " x " << K << std::endl;
    std::cout << "Image Dimension: " << B << " x " << C << " x " << H << " x " << W << std::endl;
    std::cout << "Output Dimension: " << B << " x " << M << " x " << H_out << " x " << W_out << std::endl;
#endif

    dim3 dimGrid(B, M, W_num * H_num);
    dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH, 1);

    // call the forward kernel iteratively to achieve the stream effect
    for (i = 0; i < nStreams; ++i) {
        conv_forward_kernel<<<dimGrid, dimBlock, 0, streams[i]>>>((*device_y_ptr)+i*y_bytes_s, (*device_x_ptr)+i*x_bytes_s, *device_k_ptr, B, M, C, H, W, K);
    }

    // copy the result back to y
    for (i = 0; i < nStreams; ++i) {
        hipMemcpyAsync(host_y + i*y_bytes_s, (*device_y_ptr) + i*y_bytes_s , y_size_s, hipMemcpyDeviceToHost, streams[i]);
    }

    hipDeviceSynchronize();

    // destroy the streams
    for (i = 0; i < nStreams; ++i) {
      hipStreamDestroy(streams[i]);
    }

    // free
    hipFree(*device_x_ptr);
    hipFree(*device_y_ptr);
    hipFree(*device_k_ptr);
    
#ifdef CONV_DEBUG
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        std::cout<<"CUDA error@conv_forward_gpu_prolog: "<<hipGetErrorString(error)<<std::endl;
        exit(-1);
    }
#endif
}


__host__ void GPUInterface::conv_forward_gpu(
    float *device_y, const float *device_x, const float *device_k, 
    const int B, const int M, const int C, const int H, const int W, const int K)
{

    return;

    // Set the kernel dimensions and call the kernel
    const int H_out = H - K + 1;
    const int W_out = W - K + 1;

    int W_num = ceil(W_out / (BLOCK_WIDTH * 1.0)),
        H_num = ceil(H_out / (BLOCK_WIDTH * 1.0));

    std::cout << "TEST NAME: " << TEST_NAME << std::endl;

#ifdef CONV_DEBUG
    // print dimension information
    std::cout << "Grid Dimension: " << B << " x " << M << " x " << W_num * H_num << std::endl;
    std::cout << "Block Dimension: " << BLOCK_WIDTH << " x " << BLOCK_WIDTH << " x " << 1 << std::endl;
    std::cout << "Kernel Dimension: " << M << " x " << C << " x " << K << " x " << K << std::endl;
    std::cout << "Image Dimension: " << B << " x " << C << " x " << H << " x " << W << std::endl;
    std::cout << "Output Dimension: " << B << " x " << M << " x " << H_out << " x " << W_out << std::endl;
#endif

    dim3 dimGrid(B, M, W_num * H_num);
    dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH, 1);

    conv_forward_kernel<<<dimGrid, dimBlock>>>(device_y, device_x, device_k, B, M, C, H, W, K);

    hipDeviceSynchronize();

#ifdef CONV_DEBUG
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        std::cout<<"CUDA error@conv_forward_gpu: "<<hipGetErrorString(error)<<std::endl;
        exit(-1);
    }
#endif
}


__host__ void GPUInterface::conv_forward_gpu_epilog(
    float *host_y, float *device_y, float *device_x, float *device_k, 
    const int B, const int M, const int C, const int H, const int W, const int K)
{
    return;

    // Copy the output back to host
    const int H_out = H - K + 1;
    const int W_out = W - K + 1;
    int y_size_s = B*M*H_out*W_out*sizeof(float);
        // x_size_s = B*C*H*W*sizeof(float), 
        // k_size = M*C*K*K*sizeof(float);
    hipMemcpy(host_y, device_y, y_size_s, hipMemcpyDeviceToHost);
    // Free device memory
    hipFree(device_y);
    hipFree(device_x);
    hipFree(device_k);
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}
